#include "hip/hip_runtime.h"
/*
 * Medical Image Registration ToolKit (MIRTK)
 *
 * Copyright 2016 Imperial College London
 * Copyright 2016 Andreas Schuh
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "calculate-median-image.cuh"

#include "thrust/device_ptr.h"
#include "thrust/device_malloc.h"
#include "thrust/device_free.h"
#include "thrust/device_vector.h"
#include "thrust/sequence.h"
#include "thrust/for_each.h"
#include "thrust/execution_policy.h"

#include "mirtk/Profiling.h"

namespace mirtk { namespace cuda {


// =============================================================================
// 3D median filter
// =============================================================================

// -----------------------------------------------------------------------------
/// Swap values using the provided temporary buffer
template <class T>
__device__ __forceinline__ void swap(T &a, T &b, T &tmp)
{
  tmp = a, a = b, b = tmp;
}

// -----------------------------------------------------------------------------
/// Selection sort items in place
template <class T>
__device__ void SelectionSort(T *items, int begin, int end)
{
  T tmp;
  int i, j, k;
  for (i = begin; i < end; ++i) {
    k = i;
    for (j = k+1; j < end; ++j) {
      if (items[j] < items[k]) {
        k = j;
      }
    }
    if (k != i) {
      swap(items[i], items[k], tmp);
    }
  }
}

// -----------------------------------------------------------------------------
/// Insertion sort items in place
template <class T>
__device__ void InsertionSort(T *items, int begin, int end)
{
  T x;
  int i, j;
  for (i = begin + 1; i < end; ++i) {
    x = items[i];
    j = i - 1;
    while (j >= 0 && items[j] > x) {
      items[j+1] = items[j];
      --j;
    }
    items[j+1] = x;
  }
}

// -----------------------------------------------------------------------------
/// Rearranges the items in [begin, end) such that items below the
/// returned index have value less than the pivot and items above have
/// value greater or equal the pivot
template <class T>
__device__ int Partition(T *items, int begin, int end, int pivot)
{
  T         tmp;
  const T   value = items[pivot];
  const int right = end - 1;
  int       j     = begin;
  swap(items[pivot], items[right], tmp);
  for (int i = begin; i < right; ++i) {
    if (items[i] < value) {
      swap(items[i], items[j], tmp);
      ++j;
    }
  }
  swap(items[right], items[j], tmp);
  return j;
}

// Forward declaration of mutual-recursive median-of-medians function
template <class T> __device__ int MedianOfMedians(T *, int, int, int);

// -----------------------------------------------------------------------------
/// Median of three pivot selection strategy
template <class T>
__device__ int MedianOf3(T *items, int begin, int end)
{
  const int last   = end - 1;
  const int middle = (last - begin) / 2;
  const T a = items[begin];
  const T b = items[middle];
  const T c = items[last];
  if (a < b) {
    if (a < c) return begin;
    else       return last;
  } else if (a < c) {
    if (a < b) return begin;
    else       return middle;
  } else /* a >= b && a >= c */ {
    if (b < c) return middle;
    else       return last;
  }
}

// -----------------------------------------------------------------------------
/// Pivot function of median-of-medians algorithm
template <class T>
__device__ int MedianOf5(T *items, int begin, int end)
{
  int n = end - begin;
  if (n <= 5) {
    InsertionSort(items, begin, end);
    return n / 2;
  }
  T tmp;
  for (int i = begin, j, k = begin; i < end; i += 5, ++k) {
    j = min(i + 5, end);
    InsertionSort(items, i, j);
    j = (j - i) / 2;
    swap(items[j], items[k], tmp);
  }
  n   = begin + (end - begin - 1) / 10;
  end = (end - begin + 3) /  5;
  return MedianOfMedians(items, begin, end, n);
}

// -----------------------------------------------------------------------------
/// Median-of-medians algorithm
template <class T>
__device__ int MedianOfMedians(T *items, int begin, int end, int n)
{
  int pivot;
  while (end - begin > 1) {
    pivot = MedianOf5(items, begin, end);
    pivot = Partition(items, begin, end, pivot);
    if      (n == pivot) return pivot;
    else if (n  < pivot) end   = pivot;
    else                 begin = pivot + 1;
  }
  return begin;
}

// -----------------------------------------------------------------------------
/// Iterative quickselect algorithm with median of three pivot strategy
template <class T>
__device__ int QuickSelect(T *items, int begin, int end, int n)
{
  int pivot;
  while (end - begin > 1) {
    pivot = MedianOf3(items, begin, end);
    pivot = Partition(items, begin, end, pivot);
    if      (n == pivot) return pivot;
    else if (n  < pivot) end   = pivot;
    else                 begin = pivot + 1;
  }
  return begin;
}

// -----------------------------------------------------------------------------
/// Iterative quickselect algorithm with random pivot selection
template <class T>
__device__ int QuickSelect(T *items, int begin, int end, int n, const int *rand_val, int rand_num)
{
  int pivot;
  int rand_idx = 0;
  int len = end - begin;
  while (len > 1) {
    pivot = begin + rand_val[rand_idx] % len;
    pivot = Partition(items, begin, end, pivot);
    if      (n == pivot) return pivot;
    else if (n  < pivot) end   = pivot;
    else                 begin = pivot + 1;
    len = end - begin;
    if (++rand_idx == rand_num) rand_idx = 0;
  }
  return begin;
}

// -----------------------------------------------------------------------------
__device__ __forceinline__ void ind2sub(int id, int &i, int &j, int &k, int nx, int nxy)
{
  k  = id / nxy;
  id = id % nxy;
  j  = id / nx;
  i  = id % nx;
}

// -----------------------------------------------------------------------------
__device__ __forceinline__ int sub2ind(int i, int j, int k, int nx, int nxy)
{
  return i + j * nx + k * nxy;
}

// -----------------------------------------------------------------------------
/// Kernel of ND median filter with neighborhood offsets and one thread per window
template <class T, int tx, int ty, int tz, int wx, int wy, int wz, SortingAlgorithm algo>
__global__ void MedianKernel(T* output, const T *input, const int *offsets,
                             int nx, int ny, int nz, int z0,
                             const int *d_rand = nullptr, int nrand = 0)
{
  enum {
    rx     = wx / 2,
    ry     = wy / 2,
    rz     = wz / 2,
    wn     = wx * wy * wz,
    sx     = tx,// + wx - 1,
    sy     = ty,// + wy - 1,
    sz     = tz,// + wz - 1,
    sxy    = sx * sy,
    sn     = sx * sy * sz,       // no. of subvolume elements incl. halo
    tn     = tx * ty * tz,       // no. of worker threads in block
    stride = (sn + tn - 1) / tn, // stride between local values to load
  };

  const int nxy = nx * ny;
  const int nv  = nxy * nz;

  const int bid = blockIdx .x + gridDim .x * (blockIdx .y + gridDim .y * (blockIdx.z + z0));
  const int tid = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
  const int cid = bid * blockDim.x * blockDim.y * blockDim.z + tid;

  if (cid < nv) {
    // Collectively load subvolume with coalescing into shared memory.
    // As the windows of the threads overlap, this should increase bandwidth.
    #if 0
      const int i0 = blockIdx.x;
      const int j0 = blockIdx.y * gridDim.x;
      const int k0 = blockIdx.z * gridDim.x * gridDim.y;

      int vid, vi, vj, vk; // indices into loaded subvolume in shared memory
      int gid, gi, gj, gk; // indices into input volume in global memory
      int wid;             // linear index into local window values array

      __shared__ T volume[sn];
      for (vid = tid; vid < sn; vid += stride) {
        ind2sub(vid, vi, vj, vk, sx, sxy);
        gi = i0 + vi;// - rx;
        gj = j0 + vj;// - ry;
        gk = k0 + vk;// - rz;
        gid = sub2ind(gi, gj, gk, nx, nxy);
        //if (gid < 0 || gid >= nv) volume[vid] = T(0);
        //else                      volume[vid] = input[gid];
      }
      __syncthreads();

      T window[wn];
      for (wid = 0; wid < wn; ++wid) {
        gid = cid + offsets[wid];
        if (gid < 0 || gid >= nv) {
          window[wid] = T(0);
        } else {
          ind2sub(gid, gi, gj, gk, nx, nxy);
          vi = gi - i0;// + rx;
          vj = gj - j0;// + ry;
          vk = gk - k0;// + rz;
          vid = sub2ind(vi, vj, vk, sx, sxy);
          window[wid] = volume[vid];
        }
      }
    #else
      T window[wn];
      for (int wid = 0, gid; wid < wn; ++wid) {
        gid = cid + offsets[wid];
        if (gid < 0 || gid >= nv) window[wid] = T(0);
        else                      window[wid] = input[gid];
      }
    #endif

    switch (algo) {
      case SA_InsertionSort: {
        InsertionSort(window, 0, wn);
        output[cid] = window[wn/2];
      } break;
      case SA_SelectionSort: {
        SelectionSort(window, 0, wn);
        output[cid] = window[wn/2];
      } break;
      case SA_QuickSelect_RandPivot: {
        output[cid] = window[QuickSelect(window, 0, wn, wn/2, d_rand, nrand)];
      } break;
      case SA_QuickSelect_MedianOf3: {
        output[cid] = window[QuickSelect(window, 0, wn, wn/2)];
      } break;
      case SA_MedianOfMedians: {
        output[cid] = window[MedianOfMedians(window, 0, wn, wn/2)];
      } break;
    }
  }
}

// -----------------------------------------------------------------------------
/// Launch median filter kernel for a given total no. of elements in each window
template <class TVoxel, int wx, int wy, int wz>
__host__ void RunMedianFilter(TVoxel *d_out, const TVoxel *d_in, const int *d_offsets,
                              int nx, int ny, int nz, SortingAlgorithm algo)
{
  // launch parameters
  const int nslices  = 8;
  const int nbatches = (nz + nslices - 1) / nslices;
  const int tx = 16, ty = 16, tz = 1;
  dim3 blockDim = dim3(tx, ty, tz);
  dim3 gridDim  = dim3(DivUp<unsigned int>(nx, blockDim.x),
                       DivUp<unsigned int>(ny, blockDim.y),
                       DivUp<unsigned int>(nz, blockDim.z));
  // generate pool of random numbers for quickselect
  int n_rand = 0, *h_rand = nullptr, *d_rand = nullptr;
  if (algo == SA_QuickSelect_RandPivot) {
    n_rand = wx * wy * wz;
    h_rand = new int[n_rand];
    for (int i = 0; i < n_rand; ++i) h_rand[i] = rand();
    CudaSafeCall(hipMalloc(&d_rand, n_rand * sizeof(int)));
    CudaSafeCall(hipMemcpy(d_rand, h_rand, n_rand * sizeof(int), hipMemcpyHostToDevice));
  }
  // run kernel in batches of slices to avoid kernel timeout
  MIRTKCU_START_TIMING();
  for (int batch = 0, z0 = 0; batch < nbatches; ++batch, z0 += nslices) {
    gridDim.z = min(nslices, nz - z0);
    switch (algo) {
      case SA_SelectionSort: {
        MedianKernel<TVoxel, tx, ty, tz, wx, wy, wz, SA_SelectionSort><<<gridDim, blockDim>>>(d_out, d_in, d_offsets, nx, ny, nz, z0);
      } break;
      case SA_InsertionSort: {
        MedianKernel<TVoxel, tx, ty, tz, wx, wy, wz, SA_InsertionSort><<<gridDim, blockDim>>>(d_out, d_in, d_offsets, nx, ny, nz, z0);
      } break;
      case SA_QuickSelect_RandPivot: {
        MedianKernel<TVoxel, tx, ty, tz, wx, wy, wz, SA_QuickSelect_RandPivot><<<gridDim, blockDim>>>(d_out, d_in, d_offsets, nx, ny, nz, z0, d_rand, n_rand);
      } break;
      case SA_QuickSelect_MedianOf3: {
        MedianKernel<TVoxel, tx, ty, tz, wx, wy, wz, SA_QuickSelect_MedianOf3><<<gridDim, blockDim>>>(d_out, d_in, d_offsets, nx, ny, nz, z0);
      } break;
      case SA_MedianOfMedians: {
        MedianKernel<TVoxel, tx, ty, tz, wx, wy, wz, SA_MedianOfMedians><<<gridDim, blockDim>>>(d_out, d_in, d_offsets, nx, ny, nz, z0);
      } break;
    }
    CudaSafeCall(hipPeekAtLastError());
  }
  MIRTKCU_END_TIMING("median filter kernel");
  if (d_rand) CudaWarnCall(hipFree(d_rand));
  if (h_rand) delete[] h_rand;
}

#define INSTANTIATE_FOR_VOXEL_TYPE(type) \
  template __host__ void RunMedianFilter<type,  3, 3,  1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type,  5, 5,  1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type,  7, 7,  1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 11, 11, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 13, 13, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 15, 15, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 17, 17, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 19, 19, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 21, 21, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 23, 23, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 25, 25, 1>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  \
  template __host__ void RunMedianFilter<type,  3,  3,  3>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type,  5,  5,  5>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type,  7,  7,  7>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 11, 11, 11>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 13, 13, 13>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 15, 15, 15>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 17, 17, 17>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 19, 19, 19>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 21, 21, 21>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 23, 23, 23>(type *, const type *, const int *, int, int, int, SortingAlgorithm); \
  template __host__ void RunMedianFilter<type, 25, 25, 25>(type *, const type *, const int *, int, int, int, SortingAlgorithm)
INSTANTIATE_FOR_VOXEL_TYPE(short);
INSTANTIATE_FOR_VOXEL_TYPE(unsigned short);
INSTANTIATE_FOR_VOXEL_TYPE(int);
INSTANTIATE_FOR_VOXEL_TYPE(unsigned int);
INSTANTIATE_FOR_VOXEL_TYPE(float);
INSTANTIATE_FOR_VOXEL_TYPE(double);
#undef INSTANTIATE_FOR_VOXEL_TYPE

// =============================================================================
// Median image filter using Thrust
// =============================================================================

// -----------------------------------------------------------------------------
template <class T>
struct ThrustMedianKernel
{
  const T   *_Input;
  T         *_Output;
  const int *_Offsets;
  T         *_Buffer;
  int        _SliceOffset;
  int        _NumVoxels;
  int        _NumOffsets;

  __device__ T operator ()(int i) const
  {
    T *values = _Buffer + i * _NumOffsets;
    for (int j = 0, idx; j < _NumOffsets; ++j) {
      idx = _SliceOffset + i + _Offsets[j];
      values[j] = ((idx < 0 || idx >= _NumVoxels) ? T(0) : _Input[idx]);
    }
    thrust::sort(thrust::device, values, values + _NumOffsets);
    return values[_NumOffsets/2];
  }
};

// -----------------------------------------------------------------------------
template <class T>
__host__ void RunThrustMedianFilter(T *d_out, const T *d_in,
                                    int nx, int ny, int nz,
                                    int wx, int wy, int wz)
{
  const int nslices = 1;

  const int rx = wx / 2;
  const int ry = wy / 2;
  const int rz = wz / 2;
  const int wn = wx * wy * wz;

  // calculate neighborhood offsets
  int idx = 0;
  thrust::device_vector<int> offsets(wx * wy * wz);
  for (int k = -rz; k <= rz; ++k)
  for (int j = -ry; j <= ry; ++j)
  for (int i = -rx; i <= rx; ++i, ++idx) {
    offsets[idx] = i + nx * (j + ny * k);
  }

  // median filter with buffer for unwrapped slice
  thrust::device_vector<T> buffer(nslices * nx * ny * wn);

  ThrustMedianKernel<T> kernel;
  kernel._Input      = d_in;
  kernel._Output     = d_out;
  kernel._Offsets    = thrust::raw_pointer_cast(offsets.data());
  kernel._Buffer     = thrust::raw_pointer_cast(buffer .data());
  kernel._NumVoxels  = nx * ny * nz;
  kernel._NumOffsets = wn;

  // start timer
  clock_t t_start = clock();

  // slice-by-slice due to high memory demands otherwise
  thrust::device_vector<int> seq(nslices * nx * ny);
  thrust::sequence(thrust::device, seq.begin(), seq.end(), 0);
  for (int k = 0; k < nz; k += nslices) {
    kernel._SliceOffset = k * nx * ny;
    if (k + nslices >= nz) {
      seq.resize(nslices - (k + nslices - nz + 1));
    }
    thrust::for_each(seq.begin(), seq.end(), kernel);
  }

  // wait for kernels to finish before measuring the runtime
  CudaSafeCall(hipDeviceSynchronize());
  const double t_elapsed = static_cast<double>(clock() - t_start) / CLOCKS_PER_SEC;
  PrintElapsedTime("median filter kernel [Thrust]", t_elapsed, TIME_IN_MILLISECONDS);
}

#define INSTANTIATE_FOR_VOXEL_TYPE(type) \
  template __host__ void RunThrustMedianFilter(type *, const type *, int, int, int, int, int, int)
INSTANTIATE_FOR_VOXEL_TYPE(short);
INSTANTIATE_FOR_VOXEL_TYPE(unsigned short);
INSTANTIATE_FOR_VOXEL_TYPE(int);
INSTANTIATE_FOR_VOXEL_TYPE(unsigned int);
INSTANTIATE_FOR_VOXEL_TYPE(float);
INSTANTIATE_FOR_VOXEL_TYPE(double);
#undef INSTANTIATE_FOR_VOXEL_TYPE


} } // namespace mirt::cuda
